#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file fast_top_k.cu
 * \brief Adaptation of top-k from DSSTNE
 */

// Original dsstne authors: scottlegrand, rybakov, sgkim126

#include "bitonic.h"
#include "../mshadow_op.h"
#include <mshadow/tensor.h>
#include <mshadow/expression.h>
#include <mxnet/resource.h>
#include "ordering_op-inl.h"
#include "sort_op.h"
#include <mshadow/stream_gpu-inl.h>

namespace mxnet {
namespace op {

// TODO: fwd declare dsstne implementation
// Reshape the tensors as in current implementation
// Topk
// Reshape tensors back
// OR just reshape last dimension


// CUDA macros and routines

__device__ inline uint64_t llitoulli(int64_t l)
{
    uint64_t u;
    asm("mov.b64    %0, %1;" : "=l"(u) : "l"(l));
    return u;
}

__device__ inline int64_t ullitolli(uint64_t u)
{
    int64_t l;
    asm("mov.b64    %0, %1;" : "=l"(l) : "l"(u));
    return l;
}

// Handle arbitrary API churn from new and improved thread within thread model
#if (CUDA_VERSION >= 9000)
#define SHFL(x, lane) __shfl_sync(0xffffffff, (x), (lane))
#define BALLOT(predicate) __ballot_sync(0xffffffff, (predicate))
#define ANY(predicate) __any_sync(0xffffffff, (predicate))
#else
#define SHFL(x, lane) __shfl((x), (lane))
#define BALLOT(predicate) __ballot(predicate)
#define ANY(predicate) __any(predicate)
#endif // CUDA_VERSION >= 9000

#define LAUNCHERROR(s) \
    { \
        hipError_t status = hipGetLastError(); \
        if (status != hipSuccess) { \
            printf("Error: %s launching kernel %s\n", hipGetErrorString(status), s); \
            exit(-1); \
        } \
    }
#define LAUNCHERROR_BLOCKING(s) \
    { \
        hipError_t status = hipGetLastError(); \
        if (status != hipSuccess) { \
            printf("Error: %s launching kernel %s\n", hipGetErrorString(status), s); \
            exit(-1); \
        } \
        hipDeviceSynchronize(); \
    }
#define LAUNCHERROR_NONBLOCKING(s) \
    { \
        hipError_t status = hipGetLastError(); \
        if (status != hipSuccess) { \
            printf("Error: %s launching kernel %s\n", hipGetErrorString(status), s); \
            exit(-1); \
        } \
    }

#define REDUCEERROR(error) \
    if (ANY(error != (NNFloat)0.0)) \
    { \
        uint32_t tgx            = threadIdx.x & cData._warpMask; \
        error                  += SHFL(error, tgx ^ 1); \
        error                  += SHFL(error, tgx ^ 2); \
        error                  += SHFL(error, tgx ^ 4); \
        error                  += SHFL(error, tgx ^ 8); \
        error                  += SHFL(error, tgx ^ 16); \
        if (tgx == 0) \
        { \
            atomicAdd(cData._pAccumulator, llitoulli(llrintf(ERRORSCALEF * error))); \
        } \
    }


#define REDUCE(a) \
    if (ANY((a) != (NNFloat)0.0)) \
    { \
        uint32_t tgx            = threadIdx.x & cData._warpMask; \
        a                      += SHFL((a), tgx ^ 1); \
        a                      += SHFL((a), tgx ^ 2); \
        a                      += SHFL((a), tgx ^ 4); \
        a                      += SHFL((a), tgx ^ 8); \
        a                      += SHFL((a), tgx ^ 16); \
    }

// Contains information that needs to be accessible for GPU kernels and most static hyperparameters
struct GpuData {
  unsigned int _warpSize; // Warp size
  unsigned int _warpBits; // Warp bit count
  unsigned int _warpMask; // Masks bits within a warp
  unsigned long long int* _pAccumulator; // Accumulator for error calculations
};


typedef float NNFloat;
static const float MAX_VALUE = 999999999999999.0f;
static __constant__ GpuData cData = {
    ._warpSize = 32,
    ._warpBits = 5,
    ._warpMask = 31
};

__global__ void
kCalculateTopK_32_kernel(NNFloat* pOutputBuffer, NNFloat* pKeyBuffer, uint32_t* pValueBuffer,
                         uint32_t batch, uint32_t width, uint32_t k)
{
  __shared__ volatile NNFloat sKey[64 * 4];
  __shared__ volatile uint32_t sValue[64 * 4];


//  printf("Starting ... [");
//
//  for(int i =0; i < 20; i++) {
//    printf(" %f ", pOutputBuffer[i]);
//  }
//
//  printf(" ]\n");

//  printf("cData warpmask: %d, _warpSize: %d, _warpBits: %d\n", cData._warpMask, cData._warpSize,
//         cData._warpBits);

  uint32_t pos = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
  uint32_t tgx = threadIdx.x & cData._warpMask;

  if (pos < batch) {
    NNFloat *pOutput = pOutputBuffer + pos * width;
    uint32_t offset = threadIdx.x >> cData._warpBits;
    volatile NNFloat* psKey = &sKey[64 * offset];
    volatile uint32_t* psValue = &sValue[64 * offset];

    // Initialize values to
    NNFloat k0 = -MAX_VALUE;
    NNFloat k1 = -MAX_VALUE;
    uint32_t v0 = 0;
    uint32_t v1 = 0;

    // Read first 32 elements into registers
    uint32_t wpos = tgx;
//    printf("start - pos:  %d, tgx: %d, batch: %d, width: %d, wpos: %d\n", pos, tgx, batch, width,
//           wpos );
    if (wpos < width) {
      k0 = pOutput[wpos];
      v0 = wpos;
    }
    wpos += cData._warpSize;

    // Run through remainder of data
    NNFloat minValue = -MAX_VALUE;
    uint32_t rpos = 32;
    uint32_t bufferSize = 0;
    NNFloat key1, key2;
    uint32_t value1, value2;
    uint32_t otgx;
    bool flag;
    while (rpos < width) {
      //printf("loop ...\n");
      // Read block of data
      unsigned wpos = rpos + tgx;
      NNFloat key = -MAX_VALUE;
      uint32_t value = wpos;
      if (wpos < width) {
        key = pOutput[wpos];
      }

      // Add values > minValue to shared memory buffer
      uint32_t count = BALLOT(key > minValue);
      if (key > minValue) {
        uint32_t mask = 0xffffffff >> (32 - tgx);
        uint32_t offset = __popc(count & mask);
        offset += bufferSize;
        psKey[offset] = key;
        psValue[offset] = value;
      }
      bufferSize += __popc(count);

      // Check if buffer is full
      if (bufferSize >= 32) {
        // Sort 64 elements
        k1                  = psKey[tgx];
        v1                  = psValue[tgx];
        bool flag;
        BITONICSORT64_64();

        // Shift members in shared memory to beginning
        bufferSize         -= 32;
        if (tgx < bufferSize) {
          psKey[tgx]      = psKey[tgx + 32];
          psValue[tgx]    = psValue[tgx + 32];
        }
      }

      // Advance to next block of data
      rpos                    += cData._warpSize;

      if (rpos >= width) {
//        printf("end - pos:  %d, tgx: %d, batch: %d, width: %d, wpos: %d\n", pos, tgx, batch,
//               width,  wpos );
      }
    }

    // Do final sort if buffer has any remaining data
    if ((bufferSize > 0) || (width <= 32)) {
      // Store sentinel values in registers
      k1                       = -MAX_VALUE;
      v1                       = 0;

      // Load last block of unsorted data into registers
      if (tgx < bufferSize) {
        k1                   = psKey[tgx];
        v1                   = psValue[tgx];
      }
      BITONICSORT64_64();
    }

    // Copy results to key and value pointers
    NNFloat* pKey                = pKeyBuffer + pos * k;
    uint32_t* pValue             = pValueBuffer + pos * k;
    wpos                         = tgx;
    if (wpos < k) {
      pKey[wpos]               = k0;
      pValue[wpos]             = v0;
      printf("writing to global - pos:  %d, wpos: %d, k0: %f, v0: %d\n", pos, wpos, k0, v0);
    }

    wpos                        += cData._warpSize;
  }
}

static __always_inline int fls(int x)
{
  return x ? sizeof(x) * 8 - __builtin_clz(x) : 0;
}

void kCalculateTopK(NNFloat* pOutput, NNFloat *pKey, uint32_t* pValue, uint32_t batch, uint32_t width, uint32_t k)
{
  uint32_t blocks = (batch + 3) / 4;
  if (k <= 32) {
    std::cout<<"Launching "<<blocks<<" blocks"<<std::endl;
    kCalculateTopK_32_kernel<<<blocks, 128>>>(pOutput, pKey, pValue, batch, width, k);
    LAUNCHERROR_BLOCKING("kCalculateTopK_32_kernel");
  }
  else {
    std::cout<<"Not currently supported"<<std::endl;
  }
}

/*!
 * \brief Implementation of the TopK operation
 *
 *
 * \param ctx the running context
 * \param resource temporary resource handler
 * \param src the Source blob
 * \param ret the destination blobs
 * \param k the K elements to keep
 * \param param the topk parameters
 */

void FastTopKImplGpu(mshadow::Stream<gpu>* s,
                  Resource resource,
                  const TBlob &src,
                  const std::vector<TBlob> &ret,
                  const TopKParam &param) {
  using namespace mshadow;
  using namespace mshadow::expr;
  for (auto ret_ele : ret) {
    CHECK_EQ(ret_ele.type_flag_, src.type_flag_);
  }

//  std::cout<<"ret 0 shape: "<<ret[0].shape_<<std::endl;
//  std::cout<<"ret 1 shape: "<<ret[1].shape_<<std::endl;

  // 1. Parse and initialize information
  Tensor<gpu, 1, char> workspace;
  Tensor<gpu, 1, char> temp_workspace;
  Tensor<gpu, 1, real_t> sorted_dat;
  Tensor<gpu, 1, int> indices, batch_id, sel_indices;
  Tensor<gpu, 1, NNFloat> values;
  int batch_size, element_num;  // number of batches + the size of each batch
  int axis = 0;
  bool do_transpose = false;
  bool is_ascend = false;
  int k = 0;
  TShape target_shape;
  ParseTopKParam(src.shape_, param,
                 &target_shape, &batch_size, &element_num, &axis, &k, &do_transpose, &is_ascend);
//  std::cout<<"target_shape is: "<<target_shape<<std::endl;
//  std::cout<<"src.shape_ is: "<<src.shape_<<std::endl;
//  std::cout<<"param batch_size is: "<<batch_size<<std::endl;
//  std::cout<<"element_num is: "<<element_num<<std::endl;
  //  Tensor<gpu, 3, real_t> dat = src.FlatTo3D<gpu, real_t>(axis, axis, s);
  size_t temp_size = mxnet::op::SortByKeyWorkspaceSize<int, int, gpu>(src.Size());
  temp_size = std::max(temp_size, mxnet::op::SortByKeyWorkspaceSize<int, real_t, gpu>(src.Size()));
  temp_size = std::max(temp_size, mxnet::op::SortByKeyWorkspaceSize<real_t, int, gpu>(src.Size()));
//  std::cout<<"temp_size is: "<<temp_size<<std::endl;
  size_t workspace_size = temp_size + sizeof(real_t) * src.Size() + sizeof(int) * src.Size()
      * 2 + sizeof(NNFloat) * src.Size();  // TODO: K sized?
//  std::cout<<"workspace_size is: "<<workspace_size<<std::endl;
  workspace = resource.get_space_typed<gpu, 1, char>(Shape1(workspace_size), s);
  char* workspace_curr_ptr = workspace.dptr_;
  sorted_dat = Tensor<gpu, 1, real_t>(reinterpret_cast<real_t*>(workspace_curr_ptr),
                                      Shape1(src.Size()), s);  // contain sorted dat
  workspace_curr_ptr += sizeof(real_t) * src.Size();
  indices = Tensor<gpu, 1, int>(reinterpret_cast<int*>(workspace_curr_ptr),
                                Shape1(src.Size()), s);  // indices in the original matrix
  workspace_curr_ptr += sizeof(int) * src.Size();
  batch_id = Tensor<gpu, 1, int>(reinterpret_cast<int*>(workspace_curr_ptr),
                                 Shape1(src.Size()), s);  // batch id in the original matrix
  workspace_curr_ptr += sizeof(int) * src.Size();
  values = Tensor<gpu, 1, NNFloat>(reinterpret_cast<NNFloat*>(workspace_curr_ptr),
                                   Shape1(src.Size()), s);
  workspace_curr_ptr += sizeof(NNFloat) * src.Size();


  sorted_dat = src.FlatTo1D<gpu, real_t>(s);
//  std::cout<<"Reshaped data size: "<<sorted_dat.shape_<<std::endl;
  mxnet_op::Kernel<range_fwd, gpu>::Launch(s, batch_size * element_num, 1, 0, 1,
                                           kWriteTo, indices.dptr_);

  CHECK_EQ(sorted_dat.CheckContiguous(), true);
  CHECK_EQ(indices.CheckContiguous(), true);
  temp_workspace = Tensor<gpu, 1, char>(workspace_curr_ptr, Shape1(temp_size), s);  // temp space
  workspace_curr_ptr += temp_size;
  // 2. Perform inplace batch sort using the `SortByKey` in MShadow
  // After sorting, each batch in `sorted_dat` will be sorted in the corresponding order
  //   and the `indices` will contain the corresponding index in `sorted_dat`
  // Sort the data and keep record of the correspondence to global indices.
  // Instead do a kCalculateTopK

  // kCalculateTopK(NNFloat* pOutput, NNFloat *pKey, uint32_t* pValue, uint32_t batch, uint32_t
  // width, uint32_t k)
  // Pass in ret[0] as first argument.
  kCalculateTopK(sorted_dat.dptr_,
                 values.dptr_,
                 reinterpret_cast<uint32_t*>(indices.dptr_), 32, 30000, 5);

//  mxnet::op::SortByKey(sorted_dat, indices, is_ascend, &temp_workspace);

  // Iterate over sorted_date (shape 6)
  // Calculate the corresponding batch indices of the elements
  //batch_id = indices / element_num;
  // Since the SortByKey performs stable sort, the second SortByKey will reorder
  //   the sorted_dat based on the order of the batch_id

  //mxnet::op::SortByKey(batch_id, sorted_dat, true, &temp_workspace);
  // Reorder the indices
  //batch_id = indices / element_num;
  //mxnet::op::SortByKey(batch_id, indices, true, &temp_workspace);
  // 3. Assign results to the ret blob

//  std::cout<<"We are quite happy"<<std::endl;
  if (param.ret_typ == topk_enum::kReturnIndices) {
    indices -= batch_id * element_num;
    Tensor<gpu, 2, real_t> ret_indices =
        ret[0].get_with_shape<gpu, 2, real_t>(Shape2(batch_size, k), s);
    ret_indices = tcast<real_t>(slice<1>(
        inplace_reshape(indices, Shape2(batch_size, element_num)), 0, k));

  } else {
    indices -= batch_id * element_num;
    Tensor<gpu, 2, real_t> ret_value =
        ret[0].get_with_shape<gpu, 2, real_t>(Shape2(batch_size, k), s);
    Tensor<gpu, 2, real_t> ret_indices =
        ret[1].get_with_shape<gpu, 2, real_t>(Shape2(batch_size, k), s);
    ret_value = slice<1>(inplace_reshape(values, Shape2(batch_size, element_num)), 0, k);
    ret_indices = tcast<real_t>(slice<1>(
        inplace_reshape(indices, Shape2(batch_size, element_num)), 0, k));
  }
}

void FastTopKGpu(const nnvm::NodeAttrs& attrs,
              const OpContext& ctx,
              const std::vector<TBlob>& inputs,
              const std::vector<OpReqType>& req,
              const std::vector<TBlob>& outputs) {
  const TopKParam& param = nnvm::get<TopKParam>(attrs.parsed);
  // TODO(sxjscience) We can support inplace in the future
  CHECK_EQ(req[0], kWriteTo) << "TopK does not support inplace";
//  std::cout<<"inputs size: "<<inputs.size()<<std::endl;
  FastTopKImplGpu(ctx.run_ctx.get_stream<gpu>(), ctx.requested[0], inputs[0], outputs, param);
}

NNVM_REGISTER_OP(fast_topk).set_attr<FCompute>("FCompute<gpu>", FastTopKGpu);

}  // namespace op
}  // namespace mxnet